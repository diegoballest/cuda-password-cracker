#include "hip/hip_runtime.h"
/*
	@Author Diego Alfredo Ballesteros Bautista - A01271588
	Copyright (C) 2018 Diego Alfredo Ballesteros Bautista
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include <algorithm>
#include <iostream>
#include <string>
#include <set>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define N (10*10)

__global__ void getPermutationsGPU(int len, int min, int max, char *pass, int *ans){

	*ans = 0;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	switch (len){

	case 1:
		if (tid + min == pass[0]){
			*ans = 1;
			printf("FOUND GPU: %s\n", pass);
			__threadfence();
			asm("trap;");
		}
		break;

	case 2:
		if (blockIdx.x + min == pass[0] && threadIdx.x + min == pass[1]){
			*ans = 1;
			printf("FOUND GPU: %s\n", pass);
			__threadfence();
			asm("trap;");
		}
		break;

	case 3:
		for (int c1 = min; c1 < max; c1++){
			if (c1 == pass[0] && blockIdx.x + min == pass[1] && threadIdx.x + min == pass[2]){
				*ans = 1;
				printf("FOUND GPU: %s\n", pass);
				__threadfence();
				asm("trap;");
			}
		}
		break;

	case 4:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				if (c1 == pass[0] && c2 == pass[1] && blockIdx.x + min == pass[2] && threadIdx.x + min == pass[3]){
					*ans = 1;
					printf("FOUND GPU: %s\n", pass);
					__threadfence();
					asm("trap;");
				}
			}
		}
		break;

	case 5:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					if (c1 == pass[0] && c2 == pass[1] && c3 == pass[2] && blockIdx.x + min == pass[3] && threadIdx.x + min == pass[4]){
						*ans = 1;
						printf("FOUND GPU: %s\n", pass);
						__threadfence();
						asm("trap;");
					}
				}
			}
		}
		break;

	case 6:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						if (c1 == pass[0] && c2 == pass[1] && c3 == pass[2] && c4 == pass[3] && blockIdx.x + min == pass[4] && threadIdx.x + min == pass[5]){
							*ans = 1;
							printf("FOUND GPU: %s\n", pass);
							__threadfence();
							asm("trap;");
						}
					}
				}
			}
		}
		break;

	case 7:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							if (c1 == pass[0] && c2 == pass[1] && c3 == pass[2] && c4 == pass[3] && c5 == pass[4] && blockIdx.x + min == pass[5] && threadIdx.x + min == pass[6]){
								*ans = 1;
								printf("FOUND GPU: %s\n", pass);
								__threadfence();
								asm("trap;");
							}
						}
					}
				}
			}
		}
		break;
	case 8:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							for (int c6 = min; c6 < max; c6++){
								if (c1 == pass[0] && c2 == pass[1] && c3 == pass[2] && c4 == pass[3] && c5 == pass[4] && c6 == pass[5] && blockIdx.x + min == pass[6] && threadIdx.x + min == pass[7]){
									*ans = 1;
									printf("FOUND GPU: %s\n", pass);
									__threadfence();
									asm("trap;");
								}
							}
						}
					}
				}
			}
		}
		break;
	default:
		break;
	}

}

void getPermutationsCPU(int len, int min, int max, char *pass){
	int found = 0;
	switch (len){

	case 1:
		for (int c1 = min; c1 < max; c1++){
			if (pass[0] == c1){

				found = 1;
				printf("Found CPU: %s\n", pass);
				return;
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 2:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				if (pass[0] == c1 && pass[1] == c2){
					found = 1;
					printf("Found CPU: %s\n", pass);
					return;
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 3:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3){
						found = 1;
						printf("Found CPU: %s\n", pass);
						return;
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 4:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3 && pass[3] == c4){
							found = 1;
							printf("Found CPU: %s\n", pass);
							return;
						}
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 5:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3 && pass[3] == c4 && pass[4] == c5){
								found = 1;
								printf("Found CPU: %s\n", pass);
								return;
							}
						}
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 6:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							for (int c6 = min; c6 < max; c6++){
								if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3 && pass[3] == c4 && pass[4] == c5 && pass[5] == c6){
									found = 1;
									printf("Found CPU: %s\n", pass);
									return;
								}
							}
						}
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;

	case 7:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							for (int c6 = min; c6 < max; c6++){
								for (int c7 = min; c7 < max; c7++){
									if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3 && pass[3] == c4 && pass[4] == c5 && pass[5] == c6 && pass[6] == c7){
										found = 1;
										printf("Found CPU: %s\n", pass);
										return;
									}
								}
							}
						}
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;
	case 8:
		for (int c1 = min; c1 < max; c1++){
			for (int c2 = min; c2 < max; c2++){
				for (int c3 = min; c3 < max; c3++){
					for (int c4 = min; c4 < max; c4++){
						for (int c5 = min; c5 < max; c5++){
							for (int c6 = min; c6 < max; c6++){
								for (int c6 = min; c6 < max; c6++){
									for (int c7 = min; c7 < max; c7++){
										for (int c8 = min; c8 < max; c8++){
											if (pass[0] == c1 && pass[1] == c2 && pass[2] == c3 && pass[3] == c4 && pass[4] == c5 && pass[5] == c6 && pass[6] == c7 && pass[7] == c8){
												found = 1;
												printf("Found CPU: %s\n", pass);
												return;
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
		if (found == 0){
			printf("Password not Found in CPU \n");
		}
		break;
	default:
		printf("Password too long");
		break;
	}
}

void menu(){

	printf("  /$$$$$$                  /$$                                                    \n");
	printf(" /$$__  $$                | $$                                                    \n");
	printf("| $$  \\__/ /$$   /$$  /$$$$$$$  /$$$$$$                                           \n");
	printf("| $$      | $$  | $$ /$$__  $$ |____  $$                                          \n");
	printf("| $$      | $$  | $$| $$  | $$  /$$$$$$$                                          \n");
	printf("| $$    $$| $$  | $$| $$  | $$ /$$__  $$                                          \n");
	printf("|  $$$$$$/|  $$$$$$/|  $$$$$$$|  $$$$$$$                                          \n");
	printf(" \\______/  \\______/  \\_______/ \\_______/                                          \n");
	printf(" /$$$$$$$                                                                      /$$\n");
	printf("| $$__  $$                                                                    | $$\n");
	printf("| $$  \\ $$ /$$$$$$   /$$$$$$$ /$$$$$$$ /$$  /$$  /$$  /$$$$$$   /$$$$$$   /$$$$$$$\n");
	printf("| $$$$$$$/|____  $$ /$$_____//$$_____/| $$ | $$ | $$ /$$__  $$ /$$__  $$ /$$__  $$\n");
	printf("| $$____/  /$$$$$$$|  $$$$$$|  $$$$$$ | $$ | $$ | $$| $$  \\ $$| $$  \\__/| $$  | $$\n");
	printf("| $$      /$$__  $$ \\____  $$\\____  $$| $$ | $$ | $$| $$  | $$| $$      | $$  | $$\n");
	printf("| $$     |  $$$$$$$ /$$$$$$$//$$$$$$$/|  $$$$$/$$$$/|  $$$$$$/| $$      |  $$$$$$$\n");
	printf("|__/      \\_______/|_______/|_______/  \\_____/\\___/  \\______/ |__/       \\_______/\n");
	printf("  /$$$$$$                               /$$                                       \n");
	printf(" /$$__  $$                             | $$                                       \n");
	printf("| $$  \\__/  /$$$$$$  /$$$$$$   /$$$$$$$| $$   /$$  /$$$$$$   /$$$$$$              \n");
	printf("| $$       /$$__  $$|____  $$ /$$_____/| $$  /$$/ /$$__  $$ /$$__  $$             \n");
	printf("| $$      | $$  \\__/ /$$$$$$$| $$      | $$$$$$/ | $$$$$$$$| $$  \\__/             \n");
	printf("| $$    $$| $$      /$$__  $$| $$      | $$_  $$ | $$_____/| $$                   \n");
	printf("|  $$$$$$/| $$     |  $$$$$$$|  $$$$$$$| $$ \\  $$|  $$$$$$$| $$                   \n");
	printf(" \\______/ |__/      \\_______/ \\_______/|__/  \\__/ \\_______/|__/                   \n");

	printf("\n");
	printf("1) Lower case\n");
	printf("2) Upper case\n");
	printf("3) Numbers\n");
	printf("4) All (Lower case, Upper case, Numbers and Symbols)\n");
}

int main(){

	//const int isize = N*sizeof(int);
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	int opc;
	int min, max;
	menu();
	scanf("%i", &opc);
	switch (opc){
	case 1:
		min = 97;
		max = 123;
		break;
	case 2:
		min = 65;
		max = 91;
		break;
	case 3:
		min = 48;
		max = 58;
		break;
	case 4:
		min = 33;
		max = 127;
		break;
	default:
		printf("No option found\n");
	}
	char pass[100];
	printf("Password = ");
	scanf("%s", &pass);

	int *ans = 0;
	int len = strlen(pass);

	if (strlen(pass) > 8){
		printf("\n Password too long\n");
		return 0;
	}
	printf("-----------------------\n");

	char *d_pass;
	int *d_ans;

	ans = (int*)malloc(isize);
	hipMalloc((void**)&d_pass, csize);
	hipMalloc((void**)&d_ans, isize);

	hipMemcpy(d_pass, pass, csize, hipMemcpyHostToDevice);

	//hipDeviceSetLimit(hipLimitPrintfFifoSize, 9999999999);


	clock_t timeOnGpu = clock();
	if (len == 1){
		getPermutationsGPU << <1, max - min >> > (len, min, max, d_pass, d_ans);
	}
	else {
		getPermutationsGPU << <max - min, max - min >> > (len, min, max, d_pass, d_ans);
	}

	hipMemcpy(ans, d_ans, csize, hipMemcpyDeviceToHost);

	if (*ans == 0){
		printf("Password not Found in GPU \n");
	}

	double gpuTime = (((double)clock() - timeOnGpu) / CLOCKS_PER_SEC);
	printf("Time on GPU %fs\n", gpuTime);

	printf("-----------------------\n");

	clock_t timeOnCpu = clock();
	getPermutationsCPU(len, min, max, pass);
	double cpuTime = (((double)clock() - timeOnGpu) / CLOCKS_PER_SEC);
	printf("Time on CPU %fs\n", cpuTime);

	printf("-----------------------\n");

	if (timeOnGpu < timeOnCpu){
		printf("GPU is %f times faster\n", cpuTime / gpuTime);
	}
	else if (timeOnGpu > timeOnCpu){
		printf("CPU is %f times faster\n", gpuTime / cpuTime);
	}

	free(ans);
	hipFree(d_pass);
	hipFree(d_ans);
}
